/* --- CudaDeleter ---
 * CUDA Deleter custom memory delete struct
 */

#include "CudaDeleter.h"
#include <iostream>
#include <hip/hip_runtime.h>

namespace akasha
{

namespace util
{

void CudaDeleter::operator()(void *p)
{
     //std::cerr << "NOTE: CudaDeleter: Free..." << std::endl;
     hipError_t res = hipFree(p);
     if (res != hipSuccess)
     {
         std::cerr << "ERROR: CUDA Error at delete: " << hipGetErrorString(res) << std::endl;
     }
}

}

}
